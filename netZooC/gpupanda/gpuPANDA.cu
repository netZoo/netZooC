#include "hip/hip_runtime.h"
/*
Description:
	PANDA is an algorithm that creates gene regulatory network through passing messages between gene expression networks,
	TF-Gene motif priors, and TF-TF PPI networks.

Inputs:
	-e (required) file of expression values (can alternately be a list of gene names)
	-m (required) pair file of motif edges
	-p (optional) pair file of PPI edges
	-o (optional) tag for output files
	-a (optional) value to be used for update variable, alpha (default=0.1)
	Additional options (see README): 
	-k (optional) kill the program after it has run k steps (default=1000)
	-n (optional) output a "stats" file every n steps (default, no stats file)
	-w (optional) file with list of covariate weights
	-l (optional) leave out the lth sample when building the network
	-j (optional) retain only j samples when building the network
	-r (optional) randomization options
	-s (optional) value to seed the random number generator (defaults to system time)
	-v (optional) verbose output options

Outputs:
	(tag_for_output_file)_FinalNetwork.pairs: Bipartite gene-TF regulatory network where
	each line represents an edge in the graph (a TF-GENE pair).

Examples: 
	./PANDA -e ToyExpressionData.txt -m ToyMotifData.txt -o ToyOutput
	./PANDA -e ToyExpressionData.txt -m ToyMotifData.txt -p ToyPPIData.txt -a 0.25 -o ToyOutput

Publications:
	https://doi.org/10.1371/journal.pone.0064832

Authors:
	Kimberley Glass

Changelog:
	Version 1 Modifications (May 2013):
	1) added "randomseed" variable which allows the user to specific the random number generator seed
	("srand(randomseed)").  This is useful when doing paired randomizations (e.g. if one wants the gene labels to
	be swapped the same way for two different sets of input data).
	2) added in a second method of randweights (randweight=2 set by specifying -r 3 at the command prompt).  This
	generates a random-weight value for any covariate weight that was initially greater than zero, but leaves
	zero weights unchanged.
	3) Removed criteria that a protein in the PPI must be a member of the regulatory prior.  Instead add in any
	"new" proteins into the regulatory prior assuming no known regulatory interactions.
	4) Removed criteria that the TF/motif in the regulatory prior must also be a gene in the expression data. 
	This allows regulators to take names that aren't gene names (e.g. a regulator could be TAL1::GATA1, but the
	genes are in RefSeq annotation).  One limitation is that correlation in expresion levels between "TFs" and
	genes is no longer calculated.  This information, however, was never used by PANDA, so removing the
	calculation had the added benefit of freeing up memory.
	5) added in another verboseoutput option.  Now setting "-v 2" at the command prompt will cause PANDA to print
	out additional files recording the initial and final protein-interaction and co-regulatory networks.  Changed
	the behavior of the code such that an initial regulatory network is only printed out when using this option.
	6) Increased the number of Regulators allowed by Program to 1000.
	7) Modified function that reads in regulatory and co-regulating information such that it can handle
	multiple instances.  Single instances in the input files will result in the initial value being set equal to
	the value in the "weight" column.  If an interaction is multiply defined, the initial value will be set
	equal to the sum of values associated with these instances in the "weight" column.  Undefined instances
	are given a default value of 0.

	Version 2 Modifications (July 2014):
	1) added in "LeaveOutSample" to leave out a single sample from the network reconstruction.
	2) fixed missing string termination that could cause the terminal window to become bold.
	3) added in the "JackKnife" option to designate number of samples to use in a jack-knife network.
	4) modified the ReadInExpression function to allow users to add additional rows to their expression file (likely
	header rows), so long as the first character in these rows is a hashtag (#).
	5) Changed length of "TF" in regulation struct to be 64 characters, in preparation to longer miRNA names.
	6) Created default value for outtag so that the -o command-line option is now optional instead of required.
	7) Defined MAXGENES, MAXTFS, MAXCONDITIONS, BUFSIZE, and MAXPATHLENGTH to allow easier manipulation of these
	values should they need to be altered.
	8) Added some additional outputs that tell the user what the code is doing.
	9) Put in catch when normalizing the prior for the case where a TF and its potential target gene both have no
	targets/inputs (variance of 0 in the prior).
	10) removed index, indegree and outdegree parts of the genes and regulation structures as they were not being used.
	11) Restructured code so that there are much fewer global variables and most are declared locally.
	12) Modified ReadInExpression function to sum over multiple entries of a gene in the expression file.
	13) Added in program exits triggers for if input files have more than MAXGENES MAXTFS or MAXCONDITIONS.

	Potential Future improvements:
	* change the Genes.corr and Regulation.P from symmetrix matrices to vectors to save on memory space (especially
	for the former)
	* remove the exp and stderr portions of the Genes and Regulation structs and make them local vectors (currently
	they are only used to normalize the initial PPI and corr matrices and nothing else).
	* parralize the code by adding options for multi-threading for-loops using the openmp library, shoud enhance speed.

*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <getopt.h>
#include <signal.h>
#include <string.h>
//CUDA
#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime.h"
#include ""
#include "Utilities.cuh"
#include <hipblas.h>
#include <ctype.h>

#define CR 13
#define LF 10
#define BOLD "\033[1m"
#define NORMAL "\033[0m"

#define MAXGENES 40000
#define MAXTFS 2000
#define MAXCONDITIONS 500
#define BUFSIZE 10000
#define MAXPATHLENGTH 500

FILE *fid;
FILE *statfid;
char temp[BUFSIZE];
char temp1[32];
char * temp2;

char covariate_file[MAXPATHLENGTH];
char output_file[MAXPATHLENGTH];
char outtag[MAXPATHLENGTH];
char pname[MAXPATHLENGTH];

// global variables 
double alpha;
int killstep;
int randweights;
int verboseoutput;
int weightedpearson;
int randomseed;
int NumGenes;
int NumTFs;
int NumConditions;
int NumInteractions, NumUniqueInteractions;
int noExp;
int LeaveOutSample;
int JackKnife;

// structs to hold information about the networks

typedef struct{
char name[64];			// name of regulator
double W[MAXGENES];		// Current Weight of edge;
double M[MAXGENES];		// Prior knowledge from chip-chip/motif data
double P[MAXTFS];		// PPI data
double T[MAXGENES][2];		// Temporary storage of estimates
double exp;
double stdev;
} REGULATION;
REGULATION Regulation[MAXTFS];
REGULATION d_reg[MAXTFS];

typedef struct{
char name[32];
char temp[32];
double expression[MAXCONDITIONS];
double exp;			// expected value of expression across conditions
double stdev;			// standard deviation of expression across conditions
double corr[MAXGENES];
} GENES;
GENES Genes[MAXGENES];
GENES d_gen[MAXGENES];

// functions
int Initialize(REGULATION *reg, GENES *gen);
int ReadInExpressionData(GENES *gen, char filename[]);
int ReadInPriorData(REGULATION *reg, GENES *gen, char filename[]);
int ReadInInteractionData(REGULATION *reg, GENES *gen, char filename[]);
int NormalizePriorData(REGULATION *reg, GENES *gen);
int Correlation(REGULATION *reg, GENES *gen);
int IdentityPPI(REGULATION *reg, GENES *gen);
int IdentityCorrelation(GENES *gen);

int PrintStats(REGULATION *reg, GENES *gen, char filename[]);
int PrintCoReg(REGULATION *reg, GENES *gen, char filename[]);
int PrintPPI(REGULATION *reg, GENES *gen, char filename[]);

double CDF(double value);
double inverseCDF(double value);
int RandPerm(GENES *gen);
int sort(const void *x, const void *y){if(*(const double*)y < *(const double*)x) return -1; return *(const double*)y > *(const double*)x;}
void SignalHandler(int signum);
void useage();


__global__ void LearnNetwork(REGULATION *reg, GENES *gen, int step, double hamming)
{
	// local variables
	double ExpUpdate, PPIUpdate, temphamming;
	double ExpMean, ExpStd, PPIMean, PPIStd;
	int cnt, cnt1, cnt2, d;
	double A, B, C;
	int newindex= blockIdx.x * blockDim.x + threadIdx.x;
	int stride= blockDim.x * gridDim.x;

	ExpMean=0;
	ExpStd=0;
	for(cnt1=newindex; cnt1<NumTFs; cnt1+=stride)
	{
		for(cnt2=0; cnt2<NumGenes; cnt2++)
		{
			A=0;
			B=0;
			C=0;
			d=0;
			for(cnt=0; cnt<NumGenes; cnt++)
			{
				A+=gen[cnt2].corr[cnt]*reg[cnt1].W[cnt];
				B+=gen[cnt2].corr[cnt]*gen[cnt2].corr[cnt];
				C+=reg[cnt1].W[cnt]*reg[cnt1].W[cnt];
				d++;
			}
			A=A/sqrt(B+C-fabs(A));
			
			ExpMean+=A;
			ExpStd+=A*A;
			reg[cnt1].T[cnt2][0]=A;
		}
	}
	ExpMean=ExpMean/(NumTFs*NumGenes);
	ExpStd=sqrt(ExpStd/(NumGenes*NumTFs)-ExpMean*ExpMean);
	if(verboseoutput==1) fprintf(stderr, "%f,%f;", ExpMean, ExpStd);

	PPIMean=0;
	PPIStd=0;
	for(cnt2=newindex; cnt2<NumGenes; cnt2+=stride)
	{
		for(cnt1=0; cnt1<NumTFs; cnt1++)
		{
			A=0;
			B=0;
			C=0;
			d=0;
			for(cnt=0; cnt<NumTFs; cnt++)
			{
				A+=reg[cnt].P[cnt1]*reg[cnt].W[cnt2];
				B+=reg[cnt].P[cnt1]*reg[cnt].P[cnt1];
				C+=reg[cnt].W[cnt2]*reg[cnt].W[cnt2];

			}
			A=A/sqrt(B+C-fabs(A));

			PPIMean+=A;
			PPIStd+=A*A;
			reg[cnt1].T[cnt2][1]=A;
		}
	}
	PPIMean=PPIMean/(NumTFs*NumGenes);
	PPIStd=sqrt(PPIStd/(NumGenes*NumTFs)-PPIMean*PPIMean);
	if(verboseoutput==1) fprintf(stderr, "%f,%f;", PPIMean, PPIStd);

	ExpUpdate=0.5;
	PPIUpdate=0.5;

	temphamming=0;
	for(cnt1=newindex; cnt1<NumTFs; cnt1+=stride)
	{
		for(cnt=0; cnt<NumGenes; cnt++)
		{
			reg[cnt1].T[cnt][0]=ExpUpdate*reg[cnt1].T[cnt][0]+PPIUpdate*reg[cnt1].T[cnt][1];
			temphamming+=fabs(reg[cnt1].W[cnt]-reg[cnt1].T[cnt][0]);
			reg[cnt1].W[cnt]=(1-alpha)*reg[cnt1].W[cnt]+alpha*reg[cnt1].T[cnt][0];
		}
	}
	hamming=temphamming/(NumTFs*NumGenes);
}

__global__ void UpdateCorrelation(REGULATION *reg, GENES *gen, int step)
{
	// local variables
	int cnt, cnt1, cnt2, c;
	double A, B, C;
	double CorrMean, CorrStd;

	for(cnt=0; cnt<NumGenes; cnt++) {gen[cnt].exp=0; gen[cnt].stdev=0;}

	CorrMean=0;
	CorrStd=0;
	c=0;
	for(cnt=0; cnt<NumGenes; cnt++)
	{
		for(cnt2=cnt+1; cnt2<NumGenes; cnt2++)
		{
			A=0;
			B=0;
			C=0;
			for(cnt1=0; cnt1<NumTFs; cnt1++)
			{
				A+=reg[cnt1].W[cnt2]*reg[cnt1].W[cnt];
				B+=reg[cnt1].W[cnt]*reg[cnt1].W[cnt];
				C+=reg[cnt1].W[cnt2]*reg[cnt1].W[cnt2];
			}
			A=A/sqrt(B+C-fabs(A));
			gen[cnt].corr[cnt2]=A;
			
			CorrMean+=A;
			CorrStd+=A*A;
			c++;

			gen[cnt].exp+=A;
			gen[cnt2].exp+=A;
			gen[cnt].stdev+=A*A;
			gen[cnt2].stdev+=A*A;
		}
	}

	CorrMean=CorrMean/c;
	CorrStd=sqrt(CorrStd/c-CorrMean*CorrMean);
	// if(verboseoutput==1) fprintf(stderr, "%f,%f;", CorrMean, CorrStd);

	c=0;
	for(cnt=0; cnt<NumGenes; cnt++)
	{
		gen[cnt].exp=gen[cnt].exp/(NumGenes-1);
		gen[cnt].stdev=gen[cnt].stdev/(NumGenes-1)-gen[cnt].exp*gen[cnt].exp;
		gen[cnt].exp=(NumGenes)*(sqrt(gen[cnt].stdev))*exp(2*alpha*((double) step));
		gen[cnt].corr[cnt]=(1-alpha)*gen[cnt].corr[cnt]+alpha*gen[cnt].exp;
		for(cnt2=cnt+1; cnt2<NumGenes; cnt2++)
		{
			c++;
			gen[cnt].corr[cnt2]=(1-alpha)*gen[cnt2].corr[cnt]+alpha*(gen[cnt].corr[cnt2]);
			gen[cnt2].corr[cnt]=gen[cnt].corr[cnt2];
		}
	}
	return 0;
}

__global__ void UpdatePPI(REGULATION *reg, GENES *gen, int step)
{
	// local variables
	double PPIMean, PPIStd;
	int cnt, cnt1, cnt2, c;
	double A, B, C;

	for(cnt1=0; cnt1<NumTFs; cnt1++) {reg[cnt1].exp=0; reg[cnt1].stdev=0;}
	
	PPIMean=0;
	PPIStd=0;
	c=0;
	for(cnt1=0; cnt1<NumTFs; cnt1++)
	{
		for(cnt=cnt1+1; cnt<NumTFs; cnt++)
		{
			A=0;
			B=0;
			C=0;
			for(cnt2=0; cnt2<NumGenes; cnt2++)
			{
				A+=reg[cnt1].W[cnt2]*reg[cnt].W[cnt2];
				B+=reg[cnt1].W[cnt2]*reg[cnt1].W[cnt2];
				C+=reg[cnt].W[cnt2]*reg[cnt].W[cnt2];
			}
			A=A/sqrt(B+C-fabs(A));
			
			PPIMean+=A;
			PPIStd+=A*A;
			c++;
			
			if(cnt==cnt1) reg[cnt].P[cnt]=(1-alpha)*reg[cnt].P[cnt]+A*alpha;
			else reg[cnt1].P[cnt]=A;
			reg[cnt1].exp+=A;
			reg[cnt].exp+=A;
			reg[cnt1].stdev+=A*A;
			reg[cnt].stdev+=A*A;
		}
	}
	PPIMean=PPIMean/c;
	PPIStd=sqrt(PPIStd/c-PPIMean*PPIMean);
	// if(verboseoutput==1) fprintf(stderr,"%f,%f;", PPIMean, PPIStd);

	for(cnt1=0; cnt1<NumTFs; cnt1++)
	{
		reg[cnt1].exp=reg[cnt1].exp/(NumTFs-1);
		reg[cnt1].stdev=reg[cnt1].stdev/(NumTFs-1)-reg[cnt1].exp*reg[cnt1].exp;
		reg[cnt1].exp=(NumTFs)*(sqrt(reg[cnt1].stdev))*exp(2*alpha*((double) step));
		reg[cnt1].P[cnt1]=(1-alpha)*reg[cnt1].P[cnt1]+alpha*reg[cnt1].exp;
		for(cnt=cnt1+1; cnt<NumTFs; cnt++)
		{
			reg[cnt].P[cnt1]=(1-alpha)*reg[cnt].P[cnt1]+alpha*reg[cnt1].P[cnt];
			reg[cnt1].P[cnt]=reg[cnt].P[cnt1];
		}
	}
	return 0;
}


void copyStructureToDevice(REGULATION *reg, GENES *gen, REGULATION *d_reg, GENES *d_gen, int NumGenes, int NumTFs){
        double d_W[NumGenes], d_M[NumGenes], d_P[NumTFs], d_T[NumGenes][2];
        double d_expression, d_corr;
	char d_name[64], d_name_genes[32], d_temp[32];
	//Allocate storage for structrue
        hipMalloc(&d_reg, sizeof(REGULATION));
	hipMalloc(&d_gen, sizeof(GENES));

	//Allocate storage for each piece
        // Regulation
	hipMalloc(&d_W, NumGenes*sizeof(double));
        hipMalloc(&d_M, NumGenes*sizeof(double));
        hipMalloc(&d_P, NumTFs*sizeof(double));
        hipMalloc(&d_T, 2*NumGenes*sizeof(double));
        hipMalloc(&d_name, 64);
        //Genes
	hipMalloc(&d_expression, NumConditions*sizeof(double));
        hipMalloc(&d_corr, NumGenes*sizeof(double));
        hipMalloc(&d_name_genes, 32);
        hipMalloc(&d_temp, 32);

	//Copy variables
	//structures
	hipMemcpy(d_reg,reg,sizeof(REGULATION),hipMemcpyHostToDevice);
	hipMemcpy(d_gen,gen,sizeof(GENES),hipMemcpyHostToDevice);
	//regulation
        hipMemcpy(d_W,reg->W,NumGenes,hipMemcpyHostToDevice);
        hipMemcpy(d_M,reg->M,NumGenes,hipMemcpyHostToDevice);
        hipMemcpy(d_P,reg->P,NumTFs,hipMemcpyHostToDevice);
        hipMemcpy(d_T,reg->T,2*NumGenes,hipMemcpyHostToDevice);
        hipMemcpy(d_name,reg->name,64,hipMemcpyHostToDevice);
        //update pointers
        hipMemcpy(&(d_reg->W),&d_W,sizeof(double*),hipMemcpyHostToDevice);
        hipMemcpy(&(d_reg->M),&d_M,sizeof(double*),hipMemcpyHostToDevice);
        hipMemcpy(&(d_reg->P),&d_P,sizeof(double*),hipMemcpyHostToDevice);
        hipMemcpy(&(d_reg->T),&d_T,sizeof(double*),hipMemcpyHostToDevice);
        hipMemcpy(&(d_reg->name),&d_name,sizeof(char*),hipMemcpyHostToDevice);
	//genes
	hipMemcpy(d_expression,gen->expression,NumConditions,hipMemcpyHostToDevice);
        hipMemcpy(d_corr,gen->corr,NumGenes,hipMemcpyHostToDevice);
        hipMemcpy(d_name_genes,gen->name,32,hipMemcpyHostToDevice);
        hipMemcpy(d_temp,gen->temp,32,hipMemcpyHostToDevice);
        //update pointers
        hipMemcpy(&(d_gen->expression),&d_expression,sizeof(double*),hipMemcpyHostToDevice);
        hipMemcpy(&(d_gen->corr),&d_corr,sizeof(double*),hipMemcpyHostToDevice);
        hipMemcpy(&(d_gen->name),&d_name_genes,sizeof(char*),hipMemcpyHostToDevice);
        hipMemcpy(&(d_gen->temp),&d_temp,sizeof(char*),hipMemcpyHostToDevice);
}

int main(int argc, char *argv[])
{
	// set program defaults for global variables
	strcpy(outtag, "PANDA_prediction");
	NumGenes=6000;
	NumTFs=1000;
	NumConditions=500;
	alpha=0.1;
	noExp=1;
	LeaveOutSample=0;
	JackKnife=0;
	randweights=0;
	verboseoutput=0;
	weightedpearson=0;
	randomseed=0;

	// set local variables
	int maxstep=1000;
	int outputstep=0;
	int noPPI=1;
	int noMotif=1;
	int randlabels=0;
	double hamming;

	// variables to store file names
	char interaction_file[MAXPATHLENGTH];
	char expression_file[MAXPATHLENGTH];
	char motif_file[MAXPATHLENGTH];

	extern char *optarg;
	int errflg=2;
	int s;
	strcpy(pname, argv[0]); 

	while((s = getopt(argc, argv, "a:e:m:o:p:k:l:j:n:r:s:v:w:")) != -1)
	switch(s)
	{
		case 'e':	// file name of expression data 
			strcpy(expression_file, optarg); errflg--;
			break;
		case 'a':	// set value for alpha (default 0.1)
			alpha=atof(optarg);
			break;
		case 'm':	// pair of motif edges 
			strcpy(motif_file, optarg); errflg--; noMotif=0;
			break;
		case 'p':	//pair file of PPI
			strcpy(interaction_file, optarg);  noPPI=0;
			break;
		case 'o':	// tag for output files
			strcpy(outtag, optarg);
			break;
		case 'k':
			maxstep=atoi(optarg);
			break;
		case 'n':
			outputstep=atoi(optarg);
			break;
		case 'r':
			randlabels=atoi(optarg);
			break;
		case 's':
			randomseed=atoi(optarg);
			break;
		case 'l':
			LeaveOutSample=atoi(optarg);
			break;
		case 'j':
			JackKnife=atoi(optarg);
			break;
		case 'w':
			weightedpearson=1;
			strcpy(covariate_file, optarg);
			break;
		case 'v':
			verboseoutput=atoi(optarg);
			break;
		default:
			errflg++;
	}
	
	if (errflg)
	{
		useage();
		exit (2);
	}

	signal(SIGINT, SignalHandler);
	if(randomseed==0) randomseed=time(NULL);

	fprintf(stderr, "Reading in data.  Note that the code only allocates space for up to %u Regulators, %u Target Genes, and %u expression conditions.  If you have more than this you may need to alter the code to avoid a segfault.\n", MAXTFS, MAXGENES, MAXCONDITIONS);
	// Step (0): Initialize Values
	Initialize((REGULATION *) &Regulation, (GENES *) &Genes);
	ReadInExpressionData((GENES*) &Genes, expression_file);
	if(randlabels==1) RandPerm((GENES*) &Genes);
	if(randlabels==2) randweights=1;
	if(randlabels==3) randweights=2;
	ReadInPriorData((REGULATION*) &Regulation, (GENES*) &Genes, motif_file);	

	// Report what program thinks is going on
	fprintf(stderr, "Running PANDA using the following parameters:\n");
	fprintf(stderr, "alpha=%f\n", alpha);
	fprintf(stderr, "Data-Types Being Used include:\n");
	if(noMotif==0) fprintf(stderr, "Regulation Data\n");
	if(noPPI==0) fprintf(stderr, "Protein Interaction Data\n");
	if(noExp==0) fprintf(stderr, "Expression Data\n");
	
	// Read in data-types
	if(noPPI==0) ReadInInteractionData((REGULATION*) &Regulation, (GENES*) &Genes, interaction_file);
	else IdentityPPI((REGULATION*) &Regulation, (GENES *) &Genes);
	if(noExp==0) Correlation((REGULATION *) &Regulation, (GENES *) &Genes);
	else IdentityCorrelation((GENES *) &Genes);

	if(verboseoutput==2)
	{
		sprintf(output_file, "%s_InitialNetwork.pairs", outtag);
		PrintStats((REGULATION *) & Regulation, (GENES *) & Genes, output_file);
	}
	NormalizePriorData((REGULATION*) &Regulation, (GENES *) &Genes);
	
	// Cuda vars
	int blockSize=128;// 64 32 32
	int numBlocks=(NumTFs + blockSize-1)/blockSize;

	fprintf(stderr, "\nLearning Network!\n");
	// Learn Network!
	hamming=NumTFs*NumGenes;
	killstep=0;

	// send data to device
	copyStructureToDevice((REGULATION *) &Regulation, (GENES *) &Genes,(REGULATION *) &d_reg, (GENES *) &d_gen,NumGenes,NumTFs);
	while(hamming>1e-3 && killstep<=maxstep)
	{
		hamming=0;
		// Step (1): Learn Network
		LearnNetwork<<<numBlocks,blockSize>>>((REGULATION *) &d_reg, (GENES *) &d_gen, killstep, hamming);
		
		// Step (2): Update Correlation
		UpdateCorrelation<<<numBlocks,blockSize>>>((REGULATION *) &d_reg, (GENES *) &d_gen, killstep);
		UpdatePPI<<<numBlocks,blockSize>>>((REGULATION *) &d_reg, (GENES *) &d_gen, killstep);
		
		if(outputstep>0 && killstep % outputstep == 0)
		{
			sprintf(output_file, "%s_Step%u.stats", outtag, killstep);
			PrintStats((REGULATION *) & Regulation, (GENES *) & Genes, output_file);
		}
		fprintf(stderr, "Step#%u, hamming=%f\n", killstep, hamming);
		killstep++;
	}
	sprintf(output_file, "%s_FinalNetwork.pairs", outtag);
	PrintStats((REGULATION *) & Regulation, (GENES *) & Genes, output_file);
	
	if(verboseoutput==2)
	{
		sprintf(output_file, "%s_FinalCoReg.pairs", outtag);
		PrintCoReg((REGULATION *) & d_reg, (GENES *) & d_gen, output_file);
		sprintf(output_file, "%s_FinalPPI.pairs", outtag);
		PrintPPI((REGULATION *) & d_reg, (GENES *) & d_gen, output_file);
	}
	return 0;
}


int Initialize(REGULATION *reg, GENES *gen)
{
	int cnt,cnt1,cnt2;

	for(cnt1=0; cnt1<NumTFs; cnt1++)
	{
		for(cnt2=0; cnt2<NumGenes; cnt2++)
		{
			reg[cnt1].M[cnt2]=0;
			reg[cnt1].W[cnt2]=0;
			reg[cnt1].T[cnt2][0]=0;
			reg[cnt1].T[cnt2][1]=0;
		}
		for(cnt=0; cnt<NumTFs; cnt++) reg[cnt1].P[cnt]=0;
	}
	return 0;
}

int ReadInExpressionData(GENES *gen, char filename[])
{
	int cnt, cnt2;
	int gcnt, c;

	if((fid=fopen(filename, "r"))==NULL)
	{
		printf("ERROR OPENING EXPRESSION DATA FILE\n");
		exit(1);
	}

	fprintf(stderr, "Reading In Expression Data!\n");

	cnt=0;
	cnt2=0;
	NumGenes=0;
	while(!feof(fid))
	{
		if(fgets(temp, BUFSIZE , fid) != NULL)
		{
			if(strncmp(temp,"#",1)>0)
			{
				temp2 = strtok (temp,"\t\n");
				cnt2++;
				gcnt=NumGenes;
				for(c=0; c<NumGenes; c++)
				{
					if(strcmp(temp2,gen[c].name)==0)
					{
						gcnt=c;
						break;
					}
				}
				if(gcnt==NumGenes)
				{
					cnt=0;
					strcpy(gen[gcnt].name, temp2);
					temp2 = strtok (NULL,"\t\n");
	  				while (temp2 != NULL)
  					{
						gen[gcnt].expression[cnt]=atof(temp2);
						cnt++;
						temp2 = strtok (NULL,"\t\n");
					}
					if(gcnt==0)
					{
						NumConditions=cnt;
						if(NumConditions>MAXCONDITIONS)
						{
							printf("TOO MANY CONDITIONS DETECTED IN EXPRESSION FILE. CODE UPDATE NECESSARY TO AVOID SEG-FAULT.\n");
							exit(1);
						}
					}
					NumGenes++;
					if(NumGenes>MAXGENES)
					{
						printf("TOO MANY GENES DETECTED. CODE UPDATE NECESSARY TO AVOID SEG-FAULT.\n");
						exit(1);
					}

				}
				else
				{
					temp2 = strtok (NULL,"\t\n");
  					while (temp2 != NULL)
  					{
						gen[gcnt].expression[cnt]+=atof(temp2);
						temp2 = strtok (NULL,"\t\n");
					}
				}
			}
		}
	}
	NumGenes=NumGenes;
	fclose(fid);
	fprintf(stderr, "Num Unique Genes in Expression File:%u (%u total entries).\n", NumGenes, cnt2);

	if(NumConditions>2) noExp=0;
	else
	{
		NumConditions=0;
		fprintf(stderr, "Note that an insufficient number of conditions have been found to utilize Expression Data.  Using input file as a gene list.\n");
	}
	if(NumGenes>6000)
	{
		fprintf(stderr, "Data includes information for more than 6000 genes, allocating more space\n");
		Initialize((REGULATION *) &Regulation, (GENES *) &Genes);
	}
	return 0;
}

int ReadInPriorData(REGULATION *reg, GENES *gen, char filename[])
{
	int cnt, cnt2, c;
	double P;

	NumTFs=0;
	NumInteractions=0;
	NumUniqueInteractions=0;

	if((fid=fopen(filename, "r"))==NULL)
	{
		printf("ERROR OPENING TF-GENE FILE\n");
		exit(1);
	}

	fprintf(stderr, "Reading in Motif Data!\n");

	while(fscanf(fid, "%s\t%s\t%f", temp,temp1,&P)==3) //!feof(fid))
	{
		cnt=NumTFs;
		for(c=0; c<NumTFs; c++)
		{
			if(strcmp(temp,reg[c].name)==0)
			{
				cnt=c;
				break;
			}
		}

		cnt2=NumGenes;
		for(c=0; c<NumGenes; c++)
		{
			if(strcmp(temp1,gen[c].name)==0)
			{
				cnt2=c;
				break;
			}
		}
		if(cnt2==NumGenes)
		{
			if(strcmp(temp1, "none")==0) cnt2=-1;
			else
			{
				fprintf(stderr, "WARNING: Gene '%s' found in Regulation Data but not Gene Expression Data.  It will be added with assumed equal expression across all conditions.  If this is an error, exit the program and verify your input files.\n", temp1);
				strcpy(gen[cnt2].name, temp1);
				for(c=0; c<NumConditions; c++) gen[cnt2].expression[c]=0;
				for(c=0; c<200; c++)
				{
					reg[c].M[cnt2]=0;
					reg[c].W[cnt2]=0;
					reg[c].T[cnt2][0]=0;
					reg[c].T[cnt2][1]=0;
				}
				NumGenes++;
				if(NumGenes>MAXGENES)
				{
					printf("TOO MANY GENES DETECTED. CODE UPDATE NECESSARY TO AVOID SEG-FAULT.\n");
					exit(1);
				}
			}
		}
		
		if(cnt==NumTFs)
		{
			strcpy(reg[cnt].name, temp);
			NumTFs++;
			if(NumTFs>MAXTFS)
			{
				printf("TOO MANY REGULATORS DETECTED. CODE UPDATE NECESSARY TO AVOID SEG-FAULT.\n");
				exit(1);
			}

		}

		if(cnt2>=0)
		{
			NumInteractions++;
			if(reg[cnt].M[cnt2]==0) NumUniqueInteractions++;
			reg[cnt].M[cnt2]=reg[cnt].M[cnt2]+P;
			reg[cnt].W[cnt2]=reg[cnt].W[cnt2]+P;
		}
	}
	fclose(fid);
	return 0;
}

int NormalizePriorData(REGULATION *reg, GENES *gen)
{
	int cnt, cnt1, c;
	double A, B;
	double PriorMean, PriorStd;

	int LocConditions=NumConditions;
	if(LeaveOutSample) LocConditions--;
	else if(JackKnife) LocConditions=JackKnife;
	fprintf(stderr, "\nNetwork Data Stats:\n");
	fprintf(stderr, "NumRegulators:%u, NumGenes:%u, NumConditions:%u (%u used for Network), NumRegulatoryInteractions:%u (%u unique)\n", NumTFs, NumGenes, NumConditions, LocConditions, NumInteractions, NumUniqueInteractions);

	for(cnt1=0; cnt1<NumTFs; cnt1++)
	{
		reg[cnt1].exp=0;
		reg[cnt1].stdev=0;
		for(cnt=0; cnt<NumGenes; cnt++)
		{
			reg[cnt1].exp+=reg[cnt1].W[cnt];
			reg[cnt1].stdev+=reg[cnt1].W[cnt]*reg[cnt1].W[cnt];
		}
		reg[cnt1].exp=reg[cnt1].exp/NumGenes;
		reg[cnt1].stdev=sqrt(reg[cnt1].stdev/NumGenes-reg[cnt1].exp*reg[cnt1].exp);
	}
	
	A=0;
	B=0;
	c=0;
	for(cnt=0; cnt<NumGenes; cnt++)
	{
		gen[cnt].exp=0;
		gen[cnt].stdev=0;
		for(cnt1=0; cnt1<NumTFs; cnt1++)
		{
			A+=reg[cnt1].W[cnt];
			B+=reg[cnt1].W[cnt]*reg[cnt1].W[cnt];
			c++;
			gen[cnt].exp+=reg[cnt1].W[cnt];
			gen[cnt].stdev+=reg[cnt1].W[cnt]*reg[cnt1].W[cnt];
		}
		gen[cnt].exp=gen[cnt].exp/NumTFs;
		gen[cnt].stdev=sqrt(gen[cnt].stdev/NumTFs-gen[cnt].exp*gen[cnt].exp);
	}
	A=A/c;
	B=sqrt(B/c-A*A);
	PriorMean=A;
	PriorStd=B;
	
	for(cnt1=0; cnt1<NumTFs; cnt1++)
	{
		for(cnt=0; cnt<NumGenes; cnt++)
		{
			if(gen[cnt].stdev==0)
			{
				if(reg[cnt1].stdev==0) reg[cnt1].W[cnt]=2*(reg[cnt1].W[cnt]-PriorMean)/(sqrt(2)*PriorStd);
				else reg[cnt1].W[cnt]=(reg[cnt1].W[cnt]-reg[cnt1].exp)/(sqrt(2)*reg[cnt1].stdev)+(reg[cnt1].W[cnt]-PriorMean)/(sqrt(2)*PriorStd);

			}
			else if(reg[cnt1].stdev==0) reg[cnt1].W[cnt]=(reg[cnt1].W[cnt]-PriorMean)/(sqrt(2)*PriorStd)+(reg[cnt1].W[cnt]-gen[cnt].exp)/(sqrt(2)*gen[cnt].stdev);
			else reg[cnt1].W[cnt]=(reg[cnt1].W[cnt]-reg[cnt1].exp)/(sqrt(2)*reg[cnt1].stdev)+(reg[cnt1].W[cnt]-gen[cnt].exp)/(sqrt(2)*gen[cnt].stdev);
			// reg[cnt1].M[cnt]=reg[cnt1].W[cnt];
		}
	}
	return 0;
}

int ReadInInteractionData(REGULATION *reg, GENES *gen, char filename[])
{
	int cnt, cnt1, cnt2, c;
	double A,B;
	double P;

	if((fid=fopen(filename, "r"))==NULL)
	{
		printf("ERROR OPENING PROTEIN INTERACTION DATA FILE\n");
		exit(1);
	}

	fprintf(stderr, "Reading in Protein Interation Data!\n");

	while(fscanf(fid, "%s\t%s\t%f", temp,temp1,&P)==3)
	{
		cnt1=NumTFs;
		for(c=0; c<NumTFs; c++)
		{
			if(strcmp(temp,reg[c].name)==0)
			{
				cnt1=c;
				break;
			}
		}
		if(cnt1==NumTFs)
		{
			NumTFs++;
			if(NumTFs>MAXTFS)
			{
				printf("TOO MANY REGULATORS DETECTED. CODE UPDATE NECESSARY TO AVOID SEG-FAULT.\n");
				exit(1);
			}
			fprintf(stderr, "WARNING: Protein %s found in Interaction Data but not Regulation Data.  It will be added into Regulation matrix with no known targets.  If this is an error, exit the program and verify your input files.\n", temp);
			strcpy(reg[cnt1].name, temp);
			for(cnt=0; cnt<NumGenes; cnt++)
			{
				reg[cnt1].M[cnt]=0;
				reg[cnt1].W[cnt]=0;
				reg[cnt1].T[cnt][0]=0;
				reg[cnt1].T[cnt][1]=0;
			}
			for(cnt=0; cnt<NumTFs; cnt++) reg[cnt1].P[cnt]=0;
		}

		cnt2=NumTFs;
		for(c=0; c<NumTFs; c++)
		{
			if(strcmp(temp1,reg[c].name)==0)
			{
				cnt2=c;
				break;
			}
		}
		if(cnt2==NumTFs)
		{
			NumTFs++;
			if(NumTFs>MAXTFS)
			{
				printf("TOO MANY REGULATORS DETECTED. CODE UPDATE NECESSARY TO AVOID SEG-FAULT.\n");
				exit(1);
			}
			fprintf(stderr, "WARNING: Protein %s found in Interaction Data but not Regulation Data.  It will be added into Regulation matrix with no known targets.  If this is an error, exit the program and verify your input files.\n", temp1);
			strcpy(reg[cnt2].name, temp1);
			for(cnt=0; cnt<NumGenes; cnt++)
			{
				reg[cnt2].M[cnt]=0;
				reg[cnt2].W[cnt]=0;
				reg[cnt2].T[cnt][0]=0;
				reg[cnt2].T[cnt][1]=0;
			}
			for(cnt=0; cnt<NumTFs; cnt++) reg[cnt2].P[cnt]=0;
		}
		
		reg[cnt1].P[cnt2]=reg[cnt1].P[cnt2]+P;
		reg[cnt2].P[cnt1]=reg[cnt2].P[cnt1]+P;
	}
	fclose(fid);
	for(cnt=0; cnt<NumTFs; cnt++) reg[cnt].P[cnt]=1;

	if(verboseoutput==2)
	{
		sprintf(output_file, "%s_InitialPPI.pairs", outtag);
		fid=fopen(output_file, "w");
		for(cnt=0; cnt<NumTFs; cnt++)
		{
			for(cnt1=cnt+1; cnt1<NumTFs; cnt1++)
			{
				fprintf(fid, "%s\t%s\t%f\n", reg[cnt1].name, reg[cnt].name,reg[cnt1].P[cnt]);
			}
		}
		fclose(fid);
	}

	A=0;
	B=0;
	c=0;
	for(cnt=0; cnt<NumTFs; cnt++)
	{
		reg[cnt].exp=0;
		reg[cnt].stdev=0;
		for(cnt1=0; cnt1<NumTFs; cnt1++)
		{
			A+=reg[cnt].P[cnt1];
			B+=reg[cnt].P[cnt1]*reg[cnt].P[cnt1];
			c++;
			reg[cnt].exp+=reg[cnt].P[cnt1];
			reg[cnt].stdev+=reg[cnt].P[cnt1]*reg[cnt].P[cnt1];
		}
		reg[cnt].exp=reg[cnt].exp/NumTFs;
		reg[cnt].stdev=sqrt(reg[cnt].stdev/NumTFs-reg[cnt].exp*reg[cnt].exp);
	}
	A=A/c;
	B=sqrt(B/c-A*A);
		
	for(cnt1=0; cnt1<NumTFs; cnt1++)
	{
		for(cnt=cnt1; cnt<NumTFs; cnt++)
		{
			reg[cnt].P[cnt1]=(reg[cnt].P[cnt1]-reg[cnt].exp)/(sqrt(2)*reg[cnt].stdev)+(reg[cnt].P[cnt1]-reg[cnt1].exp)/(sqrt(2)*reg[cnt1].stdev);
			reg[cnt1].P[cnt]=reg[cnt].P[cnt1];
		}
	}
	return 0;
}

int Correlation(REGULATION *reg, GENES *gen)
{
	int cnt, cnt1, cnt2, c, v;
	double A, B, C, F;
	double P;
	double covariateweight[MAXCONDITIONS];

	// Intitalize Weights Vector
	for(cnt=0; cnt<NumConditions; cnt++) covariateweight[cnt]=1;
	if(LeaveOutSample>0)
	{
		if(LeaveOutSample>NumConditions)
		{
			fprintf(stderr, "WARNING: Sample chosen to exclude does not exist. Will use all samples to build network.\n");
			LeaveOutSample=0;
		}
		else covariateweight[LeaveOutSample-1]=0;
	}
	if(JackKnife>=NumConditions)
	{
		fprintf(stderr, "WARNING: Number of Samples Chosen to Keep in Jack-knife is greater than or equal to the total number of Samples. No samples will be removed.\n");
		JackKnife=NumConditions;
	}
	else if(JackKnife>0)
	{
		if(JackKnife<3)
		{
			fprintf(stderr, "WARNING: Number of Samples Chosen to Keep in Jack-knife is too few to calculate a correlation. Defaulting to using all samples.\n");
			JackKnife=NumConditions;
		}
		else
		{
			for(cnt=JackKnife; cnt<NumConditions; cnt++) covariateweight[cnt]=0;
			randweights=1;
			fprintf(stderr, "Choosing %u Random Samples!\n", JackKnife);
			int RandConditions[MAXCONDITIONS];
			double tempweight[MAXCONDITIONS];
			srand(randomseed);
			for(c=0; c<NumConditions; ++c)
			{
				v= rand() % (c+1);
				RandConditions[c]=RandConditions[v];
				RandConditions[v]=c;
			}
			for(c=0; c<NumConditions; c++) tempweight[c]=covariateweight[RandConditions[c]];
			for(c=0; c<NumConditions; c++) covariateweight[c]=tempweight[c];
		}
	}

	if(weightedpearson==1)
	{
		if((fid=fopen(covariate_file, "r"))==NULL)
		{
			printf("ERROR OPENING COVARIARE WEIGHT FILE, DEFAULTING TO UNWEIGHTED PEARSON\n");
		}

		cnt=0;
		while(fscanf(fid, "%f", &P)==1)
		{
			covariateweight[cnt]=P;
			cnt++;
		}
		fclose(fid);
		
		if(randweights==1)
		{
			fprintf(stderr, "Randomizing Condition Labels!\n");
			int RandConditions[MAXCONDITIONS];
			double tempweight[MAXCONDITIONS];
			srand(randomseed);
			for(c=0; c<NumConditions; ++c)
			{
				v= rand() % (c+1);
				RandConditions[c]=RandConditions[v];
				RandConditions[v]=c;
			}
			for(c=0; c<NumConditions; c++) tempweight[c]=covariateweight[RandConditions[c]];
			for(c=0; c<NumConditions; c++) covariateweight[c]=tempweight[c];
		}
		
		if(cnt!=NumConditions) fprintf(stderr, "WARNING!!! NUMBER OF ROWS IN COVARIATE FILE DO NOT MATCH NUMBER OF NUMERIC COLUMNS IN EXPRESSION FILE. YOU MAY WANT TO KILL PROGRAM AND CHECK INPUT FILES\n");
	}
	else if(randweights==1)
	{
		fprintf(stderr, "Note that the random condition-labelling option only works if you invoke the weighted-pearson option and supply a covariate file.\n");
	}

	if(randweights==2)
	{
		fprintf(stderr, "Generating Random Weights for Conditions!\n");
		A=0;
		srand(randomseed);
		for(c=0; c<NumConditions; c++)
		{
			F=((double) (rand() % 100000+1))/100000;
			if(covariateweight[c]>0) {covariateweight[c]=F; A+=F;}
		}
		sprintf(output_file, "%s_RandWeights.pairs", outtag);
		fid=fopen(output_file, "w");
		for(c=0; c<NumConditions; c++)
		{
			// covariateweight[c]=covariateweight[c]/A;
			fprintf(fid, "%g\n", covariateweight[c]);
		}
		fclose(fid);
	}

	for(cnt2=0; cnt2<NumGenes; cnt2++)
	{
		gen[cnt2].exp=0;
		gen[cnt2].stdev=0;
		c=0;
		C=0;
		for(cnt=0; cnt<NumConditions; cnt++)
		{
			if(gen[cnt2].expression[cnt]==gen[cnt2].expression[cnt]) // make sure expression data exists (not nan)
			{
				gen[cnt2].exp+=gen[cnt2].expression[cnt]*covariateweight[cnt];
				gen[cnt2].stdev+=covariateweight[cnt]*pow(gen[cnt2].expression[cnt],2);
				C+=covariateweight[cnt];
				c++;
			}
		}
		gen[cnt2].exp=gen[cnt2].exp/C;
		gen[cnt2].stdev=gen[cnt2].stdev/C-pow(gen[cnt2].exp,2);
		if(gen[cnt2].stdev==0) fprintf(stderr, "No variation of expression for %s, will give default correlation of zero to all its interactions.\n", gen[cnt2].name);
	}

	for(cnt1=0; cnt1<NumGenes; cnt1++)
	{
		for(cnt2=cnt1; cnt2<NumGenes; cnt2++)
		{
			if(cnt2==cnt1) gen[cnt1].corr[cnt1]=1;
			else
			{
				A=0;
				B=0;
				C=0;
				c=0;
				for(cnt=0; cnt<NumConditions; cnt++)
				{
					if((gen[cnt2].expression[cnt]==gen[cnt2].expression[cnt] && gen[cnt1].expression[cnt]==gen[cnt1].expression[cnt]))
					{
						A+=covariateweight[cnt]*(gen[cnt1].expression[cnt]-gen[cnt1].exp)*(gen[cnt2].expression[cnt]-gen[cnt2].exp);
						B+=covariateweight[cnt]*pow(gen[cnt1].expression[cnt]-gen[cnt1].exp,2);
						C+=covariateweight[cnt]*pow(gen[cnt2].expression[cnt]-gen[cnt2].exp,2);
						c++;
					}
				}
				if(c>2 && gen[cnt1].stdev>0 && gen[cnt2].stdev>0) gen[cnt1].corr[cnt2]=A/(sqrt(B)*sqrt(C));
				else gen[cnt1].corr[cnt2]=0;
				gen[cnt2].corr[cnt1]=gen[cnt1].corr[cnt2];
			}
		}
	}

	if(verboseoutput==2)
	{
		sprintf(output_file, "%s_InitialCoReg.pairs", outtag);
		fid=fopen(output_file, "w");
		for(cnt=0; cnt<NumGenes; cnt++)
		{
			for(cnt1=cnt+1; cnt1<NumGenes; cnt1++)
			{
				fprintf(fid, "%s\t%s\t%f\n", gen[cnt].name, gen[cnt1].name,gen[cnt1].corr[cnt]);
			}
		}
		fclose(fid);
	}

	// Recast Co-regulation in Z-score space
	A=0;
	B=0;
	cnt=0;
	for(cnt1=0; cnt1<NumGenes; cnt1++)
	{
		gen[cnt1].exp=0;
		gen[cnt1].stdev=0;
		for(cnt2=0; cnt2<NumGenes; cnt2++)
		{
			A+=gen[cnt1].corr[cnt2];
			B+=gen[cnt1].corr[cnt2]*gen[cnt1].corr[cnt2];
			cnt++;
			gen[cnt1].exp+=gen[cnt1].corr[cnt2];
			gen[cnt1].stdev+=gen[cnt1].corr[cnt2]*gen[cnt1].corr[cnt2];
		}
		gen[cnt1].exp=gen[cnt1].exp/NumGenes;
		gen[cnt1].stdev=sqrt(gen[cnt1].stdev/NumGenes-gen[cnt1].exp*gen[cnt1].exp);
	}
	A=A/cnt;
	B=sqrt(B/cnt-A*A);

	for(cnt1=0; cnt1<NumGenes; cnt1++)
	{
		for(cnt2=cnt1; cnt2<NumGenes; cnt2++)
		{
			gen[cnt1].corr[cnt2]=(gen[cnt1].corr[cnt2]-gen[cnt1].exp)/(sqrt(2)*gen[cnt1].stdev)+(gen[cnt1].corr[cnt2]-gen[cnt2].exp)/(sqrt(2)*gen[cnt2].stdev);
			gen[cnt2].corr[cnt1]=gen[cnt1].corr[cnt2];
		}
	}
	return 0;
}

int IdentityPPI(REGULATION *reg, GENES *gen)
{
	int cnt, cnt1;

	for(cnt=0; cnt<NumTFs; cnt++) reg[cnt].P[cnt]=1;

	if(verboseoutput==2)
	{
		sprintf(output_file, "%s_InitialPPI.pairs", outtag);
		fid=fopen(output_file, "w");
		for(cnt=0; cnt<NumTFs; cnt++)
		{
			for(cnt1=cnt+1; cnt1<NumTFs; cnt1++)
			{
				fprintf(fid, "%s\t%s\t%f\n", reg[cnt1].name, reg[cnt].name,reg[cnt1].P[cnt]);
			}
		}
		fclose(fid);
	}

	for(cnt=0; cnt<NumTFs; cnt++)
	{
		reg[cnt].exp=0;
		reg[cnt].stdev=0;
		for(cnt1=0; cnt1<NumTFs; cnt1++)
		{
			reg[cnt].exp+=reg[cnt].P[cnt1];
			reg[cnt].stdev+=reg[cnt].P[cnt1]*reg[cnt].P[cnt1];
		}
		reg[cnt].exp=reg[cnt].exp/NumTFs;
		reg[cnt].stdev=sqrt(reg[cnt].stdev/NumTFs-reg[cnt].exp*reg[cnt].exp);
	}
	
	for(cnt1=0; cnt1<NumTFs; cnt1++)
	{
		for(cnt=0; cnt<NumTFs; cnt++)
		{
			reg[cnt].P[cnt1]=(reg[cnt].P[cnt1]-reg[cnt].exp)/(sqrt(2)*reg[cnt].stdev)+(reg[cnt].P[cnt1]-reg[cnt1].exp)/(sqrt(2)*reg[cnt1].stdev);
		}
	}
	return 0;
}
int IdentityCorrelation(GENES *gen)
{
	int cnt, cnt1, cnt2;
	double A, B;

	for(cnt=0; cnt<NumGenes; cnt++) gen[cnt].corr[cnt]=1;

	if(verboseoutput==2)
	{
		sprintf(output_file, "%s_InitialCoReg.pairs", outtag);
		fid=fopen(output_file, "w");
		for(cnt=0; cnt<NumGenes; cnt++)
		{
			for(cnt1=cnt+1; cnt1<NumGenes; cnt1++)
			{
				fprintf(fid, "%s\t%s\t%f\n", gen[cnt].name, gen[cnt1].name,gen[cnt1].corr[cnt]);
			}
		}
		fclose(fid);
	}

	A=0;
	B=0;
	cnt=0;
	for(cnt1=0; cnt1<NumGenes; cnt1++)
	{
		gen[cnt1].exp=0;
		gen[cnt1].stdev=0;
		for(cnt2=0; cnt2<NumGenes; cnt2++)
		{
			A+=gen[cnt1].corr[cnt2];
			B+=gen[cnt1].corr[cnt2]*gen[cnt1].corr[cnt2];
			cnt++;
			gen[cnt1].exp+=gen[cnt1].corr[cnt2];
			gen[cnt1].stdev+=gen[cnt1].corr[cnt2]*gen[cnt1].corr[cnt2];
		}
		gen[cnt1].exp=gen[cnt1].exp/NumGenes;
		gen[cnt1].stdev=sqrt(gen[cnt1].stdev/NumGenes-gen[cnt1].exp*gen[cnt1].exp);
	}
	A=A/cnt;
	B=sqrt(B/cnt-A*A);

	for(cnt1=0; cnt1<NumGenes; cnt1++)
	{
		for(cnt2=0; cnt2<NumGenes; cnt2++)
		{
			gen[cnt1].corr[cnt2]=(gen[cnt1].corr[cnt2]-gen[cnt1].exp)/(sqrt(2)*gen[cnt1].stdev)+(gen[cnt1].corr[cnt2]-gen[cnt2].exp)/(sqrt(2)*gen[cnt2].stdev);
		}
	}
	return 0;
}


int PrintStats(REGULATION *reg, GENES *gen, char filename[])
{
	int cnt1, cnt2;

	statfid=fopen(filename, "w");
	for(cnt1=0; cnt1<NumTFs; cnt1++)
	{
		for(cnt2=0; cnt2<NumGenes; cnt2++)
		{
			fprintf(statfid,"%s\t%s\t%f\t%f\n",reg[cnt1].name,gen[cnt2].name,reg[cnt1].M[cnt2],reg[cnt1].W[cnt2]);
		}
	}
	fclose(statfid);
	return 0;
}

int PrintPPI(REGULATION *reg, GENES *gen, char filename[])
{
	int cnt, cnt1;
	statfid=fopen(filename, "w");
	for(cnt=0; cnt<NumTFs; cnt++)
	{
		for(cnt1=cnt+1; cnt1<NumTFs; cnt1++)
		{
			fprintf(statfid, "%s\t%s\t%f\n", reg[cnt1].name, reg[cnt].name,reg[cnt1].P[cnt]);
		}
	}
	fclose(statfid);
	return 0;
}

int PrintCoReg(REGULATION *reg, GENES *gen, char filename[])
{
	int cnt,cnt1;

	statfid=fopen(filename, "w");
	for(cnt=0; cnt<NumGenes; cnt++)
	{
		for(cnt1=cnt+1; cnt1<NumGenes; cnt1++)
		{
			fprintf(statfid, "%s\t%s\t%f\n", gen[cnt].name, gen[cnt1].name,gen[cnt1].corr[cnt]);
		}
	}
	fclose(statfid);
	return 0;
}

double CDF(double value)
{
	// CDF of a Z-Score using approximation from Abramowitz and Stegun 26.2.17

	// CDF variables
	double b0;
	double b1=0.319381530;
	double b2=-0.356563782;
	double b3=1.781477937;
	double b4=-1.821255978;
	double b5=1.330274429;

	if(value>=0)
	{
		b0=1/(1+0.2316419*value);
		value=1-0.39894228*exp(-0.5*pow(value,2))*b0*(b0*(b0*(b0*(b0*b5+b4)+b3)+b2)+b1);
	}
	else
	{
		b0=1/(1-0.2316419*value);
		value=0.39894228*exp(-0.5*pow(value,2))*b0*(b0*(b0*(b0*(b0*b5+b4)+b3)+b2)+b1);
	}
	return value;
}

double inverseCDF(double value)
{
	double c1[6] = {-3.969683028665376e+01,2.209460984245205e+02,-2.759285104469687e+02,1.383577518672690e+02,-3.066479806614716e+01,2.506628277459239e+00};
	double c2[5] = {-5.447609879822406e+01,1.615858368580409e+02,-1.556989798598866e+02,6.680131188771972e+01,-1.328068155288572e+01};
	double c3[6] = {-7.784894002430293e-03,-3.223964580411365e-01,-2.400758277161838e+00,-2.549732539343734e+00,4.374664141464968e+00,2.938163982698783e+00};
	double c4[4] = {7.784695709041462e-03,3.224671290700398e-01,2.445134137142996e+00,3.754408661907416e+00};
	double p_low=0.02425;
	double p_high=1-p_low;
	double val1, val2;
	if(value<=0) value=-40;
	else if(value>=1) value=40;
	else if(value < p_low)
	{
		val1=sqrt(-2*log(value));
		value=(((((c3[0]*val1+c3[1])*val1+c3[2])*val1+c3[3])*val1+c3[4])*val1+c3[5])/((((c4[0]*val1+c4[1])*val1+c4[2])*val1+c4[3])*val1+1);
	}
	else if(value > p_high)
	{
		val1=sqrt(-2*log(1-value));
		value=-1*(((((c3[0]*val1+c3[1])*val1+c3[2])*val1+c3[3])*val1+c3[4])*val1+c3[5])/((((c4[0]*val1+c4[1])*val1+c4[2])*val1+c4[3])*val1+1);
	}
	else
	{
		val2=value-0.5;
		val1=val2*val2;
		value=(((((c1[0]*val1+c1[1])*val1+c1[2])*val1+c1[3])*val1+c1[4])*val1+c1[5])*val2/(((((c2[0]*val1+c2[1])*val1+c2[2])*val1+c2[3])*val1+c2[4])*val1+1);
	}
	return value;
}

int RandPerm(GENES *gen)
{
	int c,v;

	fprintf(stderr, "Randomizing Gene Labels!\n");
	int RandSignature[MAXGENES];
	srand(randomseed);
	
	for(c=0; c<NumGenes; ++c)
	{
		v= rand() % (c+1);
		RandSignature[c]=RandSignature[v];
		RandSignature[v]=c;
	}

	for(c=0; c<NumGenes; c++) strcpy(gen[c].temp, gen[RandSignature[c]].name);
	for(c=0; c<NumGenes; c++) strcpy(gen[c].name, gen[c].temp);
	return 0;
}

void useage()
{
	printf ("%s Useage %s\n", BOLD, pname);
	printf ("%s\t-e (required) file of expression values (can alternately be a list of gene names)%s\n", BOLD, NORMAL);
	printf ("%s\t-m (required) pair file of motif edges%s\n", BOLD, NORMAL);
	printf ("%s\t-p (optional) pair file of PPI edges%s\n", BOLD, NORMAL);
	printf ("%s\t-o (optional) tag for output files%s\n", BOLD, NORMAL);
	printf ("%s\t-a (optional) value to be used for update variable, alpha (default=0.1)%s\n", BOLD, NORMAL);
	printf ("%s Additional options (see README): %s\n", BOLD, NORMAL);
	printf ("%s\t-k (optional) kill the program after it has run k steps (default=1000)%s\n", BOLD, NORMAL);
	printf ("%s\t-n (optional) output a \"stats\" file every n steps (default, no stats file)%s\n", BOLD, NORMAL);
	printf ("%s\t-w (optional) file with list of covariate weights%s\n", BOLD, NORMAL);
	printf ("%s\t-l (optional) leave out the lth sample when building the network%s\n", BOLD, NORMAL);
	printf ("%s\t-j (optional) retain only j samples when building the network%s\n", BOLD, NORMAL);
	printf ("%s\t-r (optional) randomization options%s\n", BOLD, NORMAL);
	printf ("%s\t-s (optional) value to seed the random number generator (defaults to system time)%s\n", BOLD, NORMAL);
	printf ("%s\t-v (optional) verbose output options%s\n", BOLD, NORMAL);
}

void SignalHandler(int signum)
{
	sprintf(output_file, "%s_Step%u.stats", outtag, killstep);
	PrintStats((REGULATION *) & Regulation, (GENES *) & Genes, output_file);
	fprintf(stderr, "Caught signal %d.  Current values printed to %s\n", signum, output_file);
	exit(signum);
}
